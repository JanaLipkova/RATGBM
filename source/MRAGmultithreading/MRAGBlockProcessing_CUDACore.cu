#include "hip/hip_runtime.h"
/*
 *  MRAGBlockProcessing_CUDA.cu
 *  MRAG
 *
 *  Created by Diego Rossinelli on 10/10/08.
 *  Copyright 2008 CSE Lab, ETH Zurich. All rights reserved.
 *
 */
 

#include <cutil.h>
#include <stdio.h>

#define _CUDA_SIDE
#include "MRAGcore/MRAGCommon.h"
#include "MRAGcore/MRAGBlock.h"
#include "MRAGcore/MRAGBoundaryBlockPack.h"
#undef _CUDA_SIDE

using namespace MRAG;
#include "MRAGBlockProcessing_CUDACore.h"

template<class B, class E, class Processing>
__global__  
void SimpleProcessing(Processing& processing, UniformPartition partition, BlockInfo * vInfo);


template< class B, class E, class Processing>
__global__
void GatheringProcessingInside(Processing& processing, UniformPartition partition, BlockInfo * vInfo);


template< class B, class E, class Processing>
__global__  void GatheringProcessingBoundary(Processing& processing, 
		UniformPartition* lutReadData, UniformPartition* lutWriteData,int nPatchesPerBlock,  
		BlockInfo * vInfo, BoundaryBlockPack** vBBPacks);
		

template<class BlockType_>
struct BlockProcessing_CUDACore						
{
	typedef BlockType_ B;
	typedef typename BlockType_::ElementType E;

	template< template <typename X> class Processing> 
	static void process_mmt(BlockInfo * vInfo,  const int nBlocks, Processing<B>& processing, Processing<B>* processingGPU, const int nThreads[3])
	{
		const int s[3] = {0,0,0};
		const int e[3] = {B::sizeX, B::sizeY, B::sizeZ};
		
		UniformPartition partition(s, e, nThreads);
		
		const int nOriginalAmount = sizeof(E)*(partition.work_per_thread[0]*partition.work_per_thread[1]*partition.work_per_thread[2])* nThreads[0]*nThreads[1]*nThreads[2];
		
		//limit SHMEM usage
		{
			const int KB = 1024;
			const int nMaxBytesSHMEM = 20*KB;
			
			
			if (nOriginalAmount>nMaxBytesSHMEM)
			{
				const bool not_one[3] = {
					partition.work_per_thread[0] > 1, 
					partition.work_per_thread[1] > 1,
					partition.work_per_thread[2] > 1
				};
				
				const int countNotOne = (int)not_one[0]+ (int)not_one[1]+ (int)not_one[2];

				const double proportionality_constant = pow(nMaxBytesSHMEM/(double)nOriginalAmount, 1./countNotOne);
				
				const int vMaximalAmountOfWork[3] = { //per thread, per pass
					not_one[0]? max(1,(int)floor(partition.work_per_thread[0]*proportionality_constant)) : 1,
					not_one[1]? max(1,(int)floor(partition.work_per_thread[1]*proportionality_constant)) : 1,
					not_one[2]? max(1,(int)floor(partition.work_per_thread[2]*proportionality_constant)) : 1
				};
				
				partition.setup(s, e, nThreads, vMaximalAmountOfWork);
			}
		}

			dim3 gridSize(nBlocks,1+0*partition.passes_CPU(),1);
		dim3 blockSize(nThreads[0]*nThreads[1]*nThreads[2], 1, 1);
		//printf("work per thread now: %d %d %d, passes %d\n", partition.work_per_thread[0], partition.work_per_thread[1], partition.work_per_thread[2], partition.passes_CPU());
		const int nSharedMemorySize = 
			(partition.work_per_thread[0]*partition.work_per_thread[1]*partition.work_per_thread[2])* 
			sizeof(E)*nThreads[0]*nThreads[1]*nThreads[2];
			
		Processing<BlockLab_CUDA<typename BlockType_::ElementType> >* labProcessing = (Processing<BlockLab_CUDA<typename BlockType_::ElementType> >*)processingGPU;
		
		//SimpleProcessing<BlockType_,  typename BlockType_::ElementType ><<< gridSize, blockSize, nSharedMemorySize >>>(*labProcessing, partition, vInfo);
		SimpleProcessing<BlockType_,  typename BlockType_::ElementType ><<< gridSize, blockSize >>>(*processingGPU, partition, vInfo);
	}
	
	template< template <typename X> class Processing> 
	static void process_mmt(BlockInfo * vInfo, BoundaryBlockPack** vBBPacks, const int nBlocks, 
		UniformPartition * ptrReadLutGPU, UniformPartition * ptrWriteLutGPU, const int nSubspaces, 
		Processing<BlockType_>& processing, Processing<BlockType_>*  processingGPU, const int nThreads[3], const int shmemsize)
	{
		//1. compute data-partitions for the inside, launch the kernels
		//2. compute data-partitions for the boundary, launch the kernels

		//1.
		{
			dim3 gridSize(nBlocks,1,1);
			dim3 blockSize(nThreads[0]*nThreads[1]*nThreads[2], 1, 1);
			
			const int inside_start[3] = {
				- processing.stencil_start[0],
				- processing.stencil_start[1],
				- processing.stencil_start[2]
			};
			
			const int inside_end[3] = {
				B::sizeX - processing.stencil_end[0]+ 1,
				B::sizeY - processing.stencil_end[1]+ 1,
				B::sizeZ - processing.stencil_end[2]+ 1
			};
			
			UniformPartition partition(inside_start, inside_end, nThreads);
			//printf("work per thread now: %d %d %d\n", partition.work_per_thread[0], partition.work_per_thread[1], partition.work_per_thread[2]);
			const int nOriginalAmountSHMEM = 
				sizeof(E)*
				(partition.work_per_thread[0]*partition.threads[0]  + processing.stencil_end[0] - 1 - processing.stencil_start[0])*
				(partition.work_per_thread[1]*partition.threads[1]  + processing.stencil_end[1] - 1 - processing.stencil_start[1])*
				(partition.work_per_thread[2]*partition.threads[2]  + processing.stencil_end[2] - 1 - processing.stencil_start[2]);
				
				//printf("AMOUNT SHMEM %f\n KB", nOriginalAmountSHMEM/1024.);

				
			//limit SHMEM usage
			{
				const int KB = 1024;
				const int nMaxBytesSHMEM = 8*KB;//16*KB;
				
				if (nOriginalAmountSHMEM>nMaxBytesSHMEM)
				{
					const bool not_one[3] = {
						partition.work_per_thread[0] > 1, 
						partition.work_per_thread[1] > 1,
						partition.work_per_thread[2] > 1
					};
					
					const int countNotOne = (int)not_one[0]+ (int)not_one[1]+ (int)not_one[2];
					
					const int nConstantAmount = 
						sizeof(E)* 
						(processing.stencil_end[0] - 1 - processing.stencil_start[0])*
						(processing.stencil_end[1] - 1 - processing.stencil_start[1])*
						(processing.stencil_end[2] - 1 - processing.stencil_start[2]);
					
					const int nVariableAmount = 
						sizeof(E)*
						(partition.work_per_thread[0]*partition.threads[0])*
						(partition.work_per_thread[1]*partition.threads[1])*
						(partition.work_per_thread[2]*partition.threads[2]);
						
					const double proportionality_constant = pow((nMaxBytesSHMEM - nConstantAmount)/(double)nVariableAmount, 1./countNotOne);
					
					const int vMaximalAmountOfWork[3] = { //per thread, per pass
						not_one[0]? max(1,(int)floor(partition.work_per_thread[0]*proportionality_constant)) : 1,
						not_one[1]? max(1,(int)floor(partition.work_per_thread[1]*proportionality_constant)) : 1,
						not_one[2]? max(1,(int)floor(partition.work_per_thread[2]*proportionality_constant)) : 1
					};
					
					partition.setup(inside_start, inside_end, nThreads, vMaximalAmountOfWork);
				}
			}
			
			const int nCorrectAmountSHMEM = 
			sizeof(E)*
			(partition.work_per_thread[0]*partition.threads[0]  + processing.stencil_end[0] - 1 - processing.stencil_start[0])*
			(partition.work_per_thread[1]*partition.threads[1]  + processing.stencil_end[1] - 1 - processing.stencil_start[1])*
			(partition.work_per_thread[2]*partition.threads[2]  + processing.stencil_end[2] - 1 - processing.stencil_start[2]);
			
			
			//printf("FIXED!\n");
			//printf("CORRECTED AMOUNT SHMEM %f\n KB", nCorrectAmountSHMEM/1024.);
			printf("work per thread now: %d %d %d, passes %d, SHMEM=%.2f KB\n", partition.work_per_thread[0], partition.work_per_thread[1], partition.work_per_thread[2], partition.passes_CPU(),nCorrectAmountSHMEM/1024.);
			//processing.dt = 0;
		
		
		//	exit(0);
			Processing<BlockLab_CUDA<typename BlockType_::ElementType> >* labProcessing = (Processing<BlockLab_CUDA<typename BlockType_::ElementType> >*)processingGPU;
			GatheringProcessingInside<BlockType_,  typename BlockType_::ElementType  ><<< gridSize, blockSize, nCorrectAmountSHMEM >>>(*labProcessing, partition, vInfo);
			//exit(0);
			
			
			//GatheringProcessingInside<BlockType_,  typename BlockType_::ElementType  ><<< gridSize, blockSize >>>(*processingGPU, partition, vInfo);
			//CUDA_SAFE_CALL(hipMemcpy( &processing,processingGPU, sizeof(Processing<BlockType_>), hipMemcpyDeviceToHost));
			//printf("TIMESTEP : %e\n", processing.dt);
		//	exit(0);
		}

		//2.
		{
			const int nPatches = nSubspaces;
			dim3 gridSize(nPatches*nBlocks,1,1);
			dim3 blockSize(nThreads[0]*nThreads[1]*nThreads[2], 1, 1);
			
			Processing<BlockLab_CUDA<typename BlockType_::ElementType> >* labProcessing = (Processing<BlockLab_CUDA<typename BlockType_::ElementType> >*)processingGPU;
			
			GatheringProcessingBoundary<BlockType_ , typename BlockType_::ElementType> <<< gridSize, blockSize, shmemsize >>>
				(*labProcessing, ptrReadLutGPU, ptrWriteLutGPU, nPatches, vInfo, vBBPacks);
			
		}
		int ii;
	//	scanf("%d\n", &ii);
		//cin>>ii;
	}
};

#define _CUDA_SIDE
#include "../DemoCompressibleFlow/CompressibleFlowTypes.h"
#undef _CUDA_SIDE
/*
template<class B, class E, class Processing>
__global__  
void SimpleProcessing(Processing& processing, UniformPartition partition, BlockInfo * vInfo)
{	
	Processing local_processing = processing;
	BlockInfo info = vInfo[blockIdx.x];
	B& block = *((B*)info.ptrBlock);
	
	int thread_index[3];
	partition.findThreadIndex(threadIdx.x, thread_index);
	
	const int pass = blockIdx.y;
	int s[3], e[3];
	if (partition.findWorkingInterval(thread_index, pass, s, e))
		local_processing(info, block, s, e);
}
*/
/*
template<class B, class E, class Processing>
__global__  
void SimpleProcessing(Processing& processing, UniformPartition partition, BlockInfo * vInfo)
{
	BlockInfo info = vInfo[blockIdx.x];
	B& block = *((B*)info.ptrBlock);
	
	Processing local_processing = processing;

	int thread_index[3];
	partition.findThreadIndex(threadIdx.x, thread_index);	
	const int nWorkPerThreadSize = 
		partition.work_per_thread[0]*
		partition.work_per_thread[1]*
		partition.work_per_thread[2];
	
	extern __shared__ float _cache[];
	E * cache = (E *)_cache;
	
	const char cache_start[3] = {0,0,0};
	const char cache_end[3] = {
		partition.work_per_thread[0],
		partition.work_per_thread[1],
		partition.work_per_thread[2]
	};
	BlockLab_CUDA<E> lab(cache_start, cache_end, cache + nWorkPerThreadSize*threadIdx.x);
	
	int s[3], e[3];
	for(int pass=0; pass<partition.passes(); pass++)
	{
		if(partition.findWorkingInterval(thread_index,pass, s, e))
		{
			int i[3];

			for(i[2]=s[2]; i[2]<e[2]; i[2]++)
			for(i[1]=s[1]; i[1]<e[1]; i[1]++)
			for(i[0]=s[0]; i[0]<e[0]; i[0]++)
				lab(i[0]-s[0], i[1]-s[1], i[2]-s[2]) = block(i[0], i[1], i[2]);
			
			 int local_start[3] = {0,0,0};
			 int local_end[3] = {
				partition.work_per_thread[0],
				partition.work_per_thread[1],
				partition.work_per_thread[2]
			};
			
			info.origin[0] += info.h[0]*s[0];
			info.origin[1] += info.h[1]*s[1];
			info.origin[2] += info.h[2]*s[2];
			
			local_processing(info, lab, local_start, local_end);
			
			info.origin[0] -= info.h[0]*s[0];
			info.origin[1] -= info.h[1]*s[1];
			info.origin[2] -= info.h[2]*s[2];
			
			for(i[2]=s[2]; i[2]<e[2]; i[2]++)
			for(i[1]=s[1]; i[1]<e[1]; i[1]++)
			for(i[0]=s[0]; i[0]<e[0]; i[0]++)
				block(i[0], i[1], i[2]) = lab(i[0]-s[0], i[1]-s[1], i[2]-s[2]);
		}
	}
}
*/

template<class B, class E, class Processing>
__global__  
void SimpleProcessing(Processing& processing, UniformPartition partition, BlockInfo * vInfo)
{	
	Processing local_processing = processing;
	BlockInfo info = vInfo[blockIdx.x];
	B& block = *((B*)info.ptrBlock);
	
	int thread_index[3];
	partition.findThreadIndex(threadIdx.x, thread_index);
	
	int s[3], e[3];
	for(int pass=0; pass<partition.passes(); pass++)
		if (partition.findWorkingInterval(thread_index, pass, s, e))
			local_processing(info, block, s, e);
}


template< class B, class E, class Processing>
__global__
void GatheringProcessingInside(Processing& processing, UniformPartition partition, BlockInfo * vInfo)
{
	Processing local_processing = processing;
	BlockInfo info = vInfo[blockIdx.x];
	B& block = *((B*)info.ptrBlock);

	int thread_index[3];
	partition.findThreadIndex(threadIdx.x, thread_index);	
			
	const int start_lab[3] = {
		processing.stencil_start[0],
		processing.stencil_start[1],
		processing.stencil_start[2] 
	};
	
	const int end_lab[3] = {
		partition.work_per_thread[0]*partition.threads[0]  + processing.stencil_end[0] - 1,
		partition.work_per_thread[1]*partition.threads[1]  + processing.stencil_end[1] - 1,
		partition.work_per_thread[2]*partition.threads[2]  + processing.stencil_end[2] - 1, 
	};
	
	extern __shared__ float _cache[];
	E * cache = (E *)_cache;
	BlockLab_CUDA<E> lab(start_lab, end_lab, (E *)cache);

	for(int pass=0; pass<partition.passes(); pass++)
	{
		int ref_s[3], ref_e[3];
		partition.findBlockWorkingArea(pass, ref_s, ref_e);

		int s[3], e[3], i[3];
		const bool bICanDoIt = partition.findWorkingInterval(thread_index,pass, s, e);

		const int is_first[3] = {
			(int)(thread_index[0] == 0),
			(int)(thread_index[1] == 0),
			(int)(thread_index[2] == 0),
		};

		const int is_last[3] = {
			(int)(s[0]<ref_e[0] && e[0]>=ref_e[0]),
			(int)(s[1]<ref_e[1] && e[1]>=ref_e[1]),
			(int)(s[2]<ref_e[2] && e[2]>=ref_e[2])
		};
		
		const int r_s[3] = {
			s[0] + is_first[0]*processing.stencil_start[0],
			s[1] + is_first[1]*processing.stencil_start[1],
			s[2] + is_first[2]*processing.stencil_start[2]
		};
		
		const int r_e[3] = {
			e[0] + is_last[0]*(processing.stencil_end[0]-1),
			e[1] + is_last[1]*(processing.stencil_end[1]-1),
			e[2] + is_last[2]*(processing.stencil_end[2]-1)
		};
			
		__syncthreads();

		if(bICanDoIt)
		{
			for(i[2]=r_s[2]; i[2]<r_e[2]; i[2]++)
			for(i[1]=r_s[1]; i[1]<r_e[1]; i[1]++)
			for(i[0]=r_s[0]; i[0]<r_e[0]; i[0]++)
				lab(i[0]-ref_s[0], i[1]-ref_s[1], i[2]-ref_s[2]) = block(i[0], i[1], i[2]);
		}
			
		const int start_processing[3] = {
			s[0]-ref_s[0], 
			s[1]-ref_s[1], 
			s[2]-ref_s[2]
		};
		
		const int end_processing[3] = {
			e[0]-ref_s[0], 
			e[1]-ref_s[1],
			e[2]-ref_s[2]
		};
			
		__syncthreads();
			
		if(bICanDoIt)
		{
			local_processing(info, lab, start_processing, end_processing);
		}
			
		if(bICanDoIt)
		{
			for(i[2]=s[2]; i[2]<e[2]; i[2]++)
			for(i[1]=s[1]; i[1]<e[1]; i[1]++)
			for(i[0]=s[0]; i[0]<e[0]; i[0]++)
				block(i[0], i[1], i[2]) = lab(i[0]-ref_s[0], i[1]-ref_s[1], i[2]-ref_s[2]) ;
		}
	}

}
/*


template< class BlockType_, class E, class Processing>
__global__
void GatheringProcessingInside(Processing& processing, UniformPartition partition, BlockInfo * vInfo)
{
	BlockInfo info = vInfo[blockIdx.x];
	BlockType_& block = *((BlockType_*)info.ptrBlock);

	int thread_index[3];
	partition.findThreadIndex(threadIdx.x, thread_index);	
	
	int s[3], e[3];
	for(int pass=0; pass<partition.passes(); pass++)
		if(partition.findWorkingInterval(thread_index,pass, s, e))
			processing(info, block, s, e);

}*/

/*
template< class BlockType_, class Processing>
__global__
void GatheringProcessingInside(Processing& processing, UniformPartition partition, BlockInfo * vInfo)
{
	BlockInfo& info = vInfo[blockIdx.x];
	BlockType_& block = *((BlockType_*)info.ptrBlock);

	int thread_index[3];
	partition.findThreadIndex(threadIdx.x, thread_index);	
	
	int s[3], e[3];
	for(int pass=0; pass<partition.passes(); pass++)
		if(partition.findWorkingInterval(thread_index,pass, s, e))
			processing(info, block, s, e);

}
*/

template< class B, class E, class Processing>
__global__  void GatheringProcessingBoundary(Processing& processing, 
		UniformPartition* lutReadData, UniformPartition* lutWriteData,int nPatchesPerBlock,  
		BlockInfo * vInfo, BoundaryBlockPack** vBBPacks)
{
	//1. setup
	//2. find thread idx, data to work on
	//3. construct the blocklab
	//4. work using the blocklab
	//5. write back the result
	
	//1.
	const int blockID = blockIdx.x / nPatchesPerBlock;
	const int patchID = blockIdx.x % nPatchesPerBlock;
	
	BlockInfo& info = vInfo[blockID];
	B& block = *((B*)info.ptrBlock);
	BoundaryBlockPack& bbpack = *vBBPacks[blockID];
	UniformPartition& read_partition = lutReadData[patchID];
	UniformPartition& write_partition = lutWriteData[patchID];
	
	int s[3], e[3];
	
	//2.
	int thread_index[3];
	read_partition.findThreadIndex(threadIdx.x, thread_index); //must be the same as write_partition.findThreadIndex
	
	//3.
	extern __shared__ float _cache[];
	E * cache = (E *)_cache;
	read_partition.getCoveredInterval(s,e);
	BlockLab_CUDA<E> lab(s, e, (E *)cache);
	
	for(int pass=0; pass<read_partition.passes(); pass++)
		if (read_partition.findWorkingInterval(thread_index, pass, s, e))
		{
			int i[3];

			for(i[2]=s[2]; i[2]<e[2]; i[2]++)
			for(i[1]=s[1]; i[1]<e[1]; i[1]++)
			for(i[0]=s[0]; i[0]<e[0]; i[0]++)
			{
				const bool bOutside =
					i[0]<0 || i[1]<0 || i[2]<0 ||
					i[0]>= B::sizeX || i[1]>= B::sizeY || i[2]>= B::sizeZ;  
				
				if (!bOutside)
					lab(i[0], i[1], i[2]) = block(i[0], i[1], i[2]);
				else
					constructGhosts<B,E>(bbpack, i, lab(i[0], i[1],i[2]));
			}
		}
	
	__syncthreads();

	//4.
	for(int pass=0; pass<write_partition.passes(); pass++)
		if (write_partition.findWorkingInterval(thread_index,pass, s, e))
			processing(info, lab, s, e);

	__syncthreads();
	
	//5.
	for(int pass=0; pass<write_partition.passes(); pass++)
		if (write_partition.findWorkingInterval(thread_index,pass, s, e))
		{
			int i[3];
			for(i[2]=s[2]; i[2]<e[2]; i[2]++)
			for(i[1]=s[1]; i[1]<e[1]; i[1]++)
			for(i[0]=s[0]; i[0]<e[0]; i[0]++)
				block(i[0], i[1], i[2]) = lab(i[0], i[1], i[2]);
		}
}
